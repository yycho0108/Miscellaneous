#include "RandManager.h"
#include <ctime>

RandManager::RandManager(){
    hiprandCreateGenerator (&rgen,HIPRAND_RNG_PSEUDO_DEFAULT );
    hiprandSetPseudoRandomGeneratorSeed (rgen ,time(0));
}

RandManager::~RandManager(){

}

void RandManager::rand(double* arr, int n){
	//TODO : get rid of this.
	hiprandGenerateUniformDouble(rgen,arr,n);
	//hiprandGenerateNormalDouble(rgen,arr,n,0.0,0.1);
}
void RandManager::randu(double* arr, int n){
	hiprandGenerateUniformDouble(rgen,arr,n);
}

void RandManager::randn(double* arr, int n, double mean, double stddev){
	if(n%2 != 0){ //odd
		double* ptr;
		hipMalloc(&ptr, (n+1)*sizeof(double));
		hiprandGenerateNormalDouble(rgen,ptr,(n+1),mean,stddev);
		hipMemcpy(arr,ptr,n*sizeof(double),hipMemcpyDeviceToDevice);
		hipFree(ptr);
	}else{

		hiprandGenerateNormalDouble(rgen,arr,n,mean,stddev);
	}
}
